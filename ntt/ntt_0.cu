#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <chrono>
typedef __int128 int128_t;

#define N 16384
#define P 4179340454199820289ULL
#define G 3

__device__ unsigned long long d_qmul(unsigned long long a, unsigned long long b, unsigned long long mod) {
    int128_t res = (int128_t)a * b;
    return (unsigned long long)(res % mod);
}

__device__ unsigned long long d_qpow(unsigned long long x, unsigned long long y, unsigned long long p) {
    unsigned long long res = 1;
    while (y) {
        if (y & 1)
            res = d_qmul(res, x, p);
        x = d_qmul(x, x, p);
        y >>= 1;
    }
    return res;
}

__global__ void bit_reverse_indices(int* r, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int log_n = log2f(n);
        int x = idx, rev = 0;
        for (int j = 0; j < log_n; j++) {
            rev = (rev << 1) | (x & 1);
            x >>= 1;
        }
        r[idx] = rev;
    }
}

__global__ void ntt_kernel(unsigned long long* x, int* r, int lim, int m, unsigned long long gn, unsigned long long p) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int k = m >> 1;
    int i = (idx / k) * m;
    int j = idx % k;

    if (idx < lim / 2) {
        unsigned long long g = d_qpow(gn, j, p);
        unsigned long long tmp = d_qmul(x[i + j + k], g, p);
        unsigned long long a = x[i + j];
        unsigned long long b = tmp;

        x[i + j + k] = (a >= b ? (a - b) : (a + p - b));
        x[i + j] = (a + b) % p;
    }
}

void ntt(std::vector<unsigned long long>& data, int n) {
    unsigned long long *d_data;
    int *d_r;

    hipMalloc(&d_data, sizeof(unsigned long long) * n);
    hipMalloc(&d_r, sizeof(int) * n);
    hipMemcpy(d_data, data.data(), sizeof(unsigned long long) * n, hipMemcpyHostToDevice);

    dim3 block_size(256);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    bit_reverse_indices<<<num_blocks, block_size>>>(d_r, n);
    hipDeviceSynchronize();

    for (int m = 2; m <= n; m <<= 1) {
        unsigned long long gn = d_qpow(G, (P - 1) / m, P);
        ntt_kernel<<<num_blocks, block_size>>>(d_data, d_r, n, m, gn, P);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(data.data(), d_data, sizeof(unsigned long long) * n, hipMemcpyDeviceToHost);
    hipFree(d_data);
    hipFree(d_r);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "NTT algorithm execution time: " << milliseconds << " milliseconds." << std::endl;
}

int main() {
    std::vector<unsigned long long> data(N, 0);

    ntt(data, N);

    return 0;
}
